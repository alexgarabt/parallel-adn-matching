#include "hip/hip_runtime.h"
/*
 * Exact genetic sequence alignment
 * (Using brute force)
 * CUDA
 */
#include <complex.h>
#include <stddef.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<limits.h>
#include<sys/time.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

/* Example of macros for error checking in CUDA */
#define CUDA_CHECK_FUNCTION( call )	{ hipError_t check = call; if ( check != hipSuccess ) fprintf(stderr, "CUDA Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }
#define CUDA_CHECK_KERNEL( )	{ hipError_t check = hipGetLastError(); if ( check != hipSuccess ) fprintf(stderr, "CUDA Kernel Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }

/* Arbitrary value to indicate that no matches are found */
#define	NOT_FOUND	-1

/* Arbitrary value to restrict the checksums period */
#define CHECKSUM_MAX	65535


/* 
 * Utils: Function to get wall time
 */
double cp_Wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/*
 * Utils: Random generator
 */
#include "rng.c"

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */
/* ADD KERNELS AND OTHER FUNCTIONS HERE */

//5-- Hacer la funcion kernel para generar la secuencia con el nuevo rng_next()
__global__ void new_generate_rng_sequence(rng_t random, float prob_G, float prob_C, float prob_A, char *seq, unsigned long length){
		// CADA HILO VA A LA POSICION DE SU ID GLOBAL
		unsigned long  tid_block = (threadIdx.x);
		unsigned block_size = (blockDim.x);
		unsigned long tid_global = (blockIdx.x * block_size + tid_block);

		//Si su tid esta fuera de rango entonces salirse y no hacer nada
		if( tid_global >= length) return;

		// Dar tantos saltos en como la posicion que te toca
		rng_skip(&random, tid_global);
		float prob = rng_next(&random);

		if( prob < prob_G) seq[tid_global] = 'G';
		else if ( prob < prob_C) seq[tid_global] = 'C';
		else if ( prob < prob_A) seq[tid_global] = 'A';
		else seq[tid_global] = 'T';
}

__global__ void search_pat_by_gpu(char *sequence, unsigned long seq_length, int pat_number, unsigned long *pat_length, char **pattern, unsigned long *pat_found, int *seq_matches){
		unsigned long  tid_block = (threadIdx.x);
		unsigned block_size = ( blockDim.x);
		unsigned long tid_global = ( blockIdx.x * block_size + tid_block);

		int pat;
		unsigned long lind;
		//Go through all the patterns
		if( tid_global < seq_length) {
			for(pat=0; pat<pat_number; pat++){

					//Check it dosent get more far than the sequence length
					if(tid_global > seq_length - pat_length[pat]) continue;

					//Search for the current pattern that belongs to this thread
					for(lind = 0; lind < pat_length[pat]; lind++){
							// The pattern doesnt match
							if(sequence[lind + tid_global] != pattern[pat][lind]) break;
					}
					//Check if the loop has ended with a match of the pattern
					if( lind == pat_length[pat] ){
							//put in pat found the min position found of the pattern
							//BECAUSE notfound is -1 and in unsigned is the biggest number possible
							atomicMin((unsigned long long *) &pat_found[pat], tid_global);
					}
			}
		}
		//Wait to all threads to start to cound seq_matches
		__syncthreads();
		if(pat_found[tid_global] == (unsigned long) NOT_FOUND) return;
		if(tid_global >= pat_number) return;

		//Now increment_matches
		for(lind=0; lind<pat_length[tid_global]; lind++){
				if(seq_matches[ pat_found[tid_global] + lind ] == (unsigned long) NOT_FOUND)
						seq_matches[pat_found[tid_global] + lind] = 0;
				else
						atomicAdd(&seq_matches[pat_found[tid_global] + lind], 1);
		}
}

__global__ void count_number_of_pat(int *return_pat_matches, unsigned long *pat_found, int pat_found_length, unsigned long *return_checksum_patfound, int seq_length, int *seq_matches, unsigned long* return_checksum_matches){
		unsigned long  tid_block = (threadIdx.x);
		unsigned block_size = ( blockDim.x);
		unsigned long tid_global = ( blockIdx.x * block_size + tid_block);

		if( tid_global < pat_found_length){
		
			if(pat_found[tid_global] != (unsigned long) NOT_FOUND){
				   	atomicAdd(return_pat_matches, 1);
					atomicAdd((unsigned long long*)return_checksum_patfound, pat_found[tid_global]);
			}
		}

		if(tid_global >= seq_length) return;
		if(seq_matches[tid_global] != (unsigned long) NOT_FOUND)
					atomicAdd((unsigned long long*)return_checksum_matches, seq_matches[tid_global]);

}


/*
 * Function: Increment the number of pattern matches on the sequence positions
 * 	This function can be changed and/or optimized by the students
 */
void increment_matches( int pat, unsigned long *pat_found, unsigned long *pat_length, int *seq_matches ) {
	unsigned long ind;	
	for( ind=0; ind<pat_length[pat]; ind++) {
		if ( seq_matches[ pat_found[pat] + ind ] == NOT_FOUND )
			seq_matches[ pat_found[pat] + ind ] = 0;
		else
			seq_matches[ pat_found[pat] + ind ] ++;
	}
}
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

/*
 * Function: Allocate new patttern
 */
char *pattern_allocate( rng_t *random, unsigned long pat_rng_length_mean, unsigned long pat_rng_length_dev, unsigned long seq_length, unsigned long *new_length ) {

	/* Random length */
	unsigned long length = (unsigned long)rng_next_normal( random, (double)pat_rng_length_mean, (double)pat_rng_length_dev );
	if ( length > seq_length ) length = seq_length;
	if ( length <= 0 ) length = 1;

	/* Allocate pattern */
	char *pattern = (char *)malloc( sizeof(char) * length );
	if ( pattern == NULL ) {
		fprintf(stderr,"\n-- Error allocating a pattern of size: %lu\n", length );
		exit( EXIT_FAILURE );
	}

	/* Return results */
	*new_length = length;
	return pattern;
}

/*
 * Function: Fill random sequence or pattern
 */
void generate_rng_sequence( rng_t *random, float prob_G, float prob_C, float prob_A, char *seq, unsigned long length) {
	unsigned long ind; 
	for( ind=0; ind<length; ind++ ) {
		double prob = rng_next( random );
		if( prob < prob_G ) seq[ind] = 'G';
		else if( prob < prob_C ) seq[ind] = 'C';
		else if( prob < prob_A ) seq[ind] = 'A';
		else seq[ind] = 'T';
	}
}

/*
 * Function: Copy a sample of the sequence
 */
void copy_sample_sequence( rng_t *random, char *sequence, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Copy sample */
	unsigned long ind; 
	for( ind=0; ind<length; ind++ )
		pattern[ind] = sequence[ind+location];
}

/*
 * Function: Regenerate a sample of the sequence
 */
void generate_sample_sequence( rng_t *random, rng_t random_seq, float prob_G, float prob_C, float prob_A, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length ) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Regenerate sample */
	rng_t local_random = random_seq;
	rng_skip( &local_random, location );
	generate_rng_sequence( &local_random, prob_G, prob_C, prob_A, pattern, length);
}


/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<seq_length> <prob_G> <prob_C> <prob_A> <pat_rng_num> <pat_rng_length_mean> <pat_rng_length_dev> <pat_samples_num> <pat_samp_length_mean> <pat_samp_length_dev> <pat_samp_loc_mean> <pat_samp_loc_dev> <pat_samp_mix:B[efore]|A[fter]|M[ixed]> <long_seed>\n");
	fprintf(stderr,"\n");
}



/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	/* 0. Default output and error without buffering, forces to write immediately */
	setbuf(stdout, NULL);
	setbuf(stderr, NULL);

	/* 1. Read scenary arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc < 15) {
		fprintf(stderr, "\n-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read argument values */
	unsigned long seq_length = atol( argv[1] );
	float prob_G = atof( argv[2] );
	float prob_C = atof( argv[3] );
	float prob_A = atof( argv[4] );
	if ( prob_G + prob_C + prob_A > 1 ) {
		fprintf(stderr, "\n-- Error: The sum of G,C,A,T nucleotid probabilities cannot be higher than 1\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}
	prob_C += prob_G;
	prob_A += prob_C;

	int pat_rng_num = atoi( argv[5] );
	unsigned long pat_rng_length_mean = atol( argv[6] );
	unsigned long pat_rng_length_dev = atol( argv[7] );
	
	int pat_samp_num = atoi( argv[8] );
	unsigned long pat_samp_length_mean = atol( argv[9] );
	unsigned long pat_samp_length_dev = atol( argv[10] );
	unsigned long pat_samp_loc_mean = atol( argv[11] );
	unsigned long pat_samp_loc_dev = atol( argv[12] );

	char pat_samp_mix = argv[13][0];
	if ( pat_samp_mix != 'B' && pat_samp_mix != 'A' && pat_samp_mix != 'M' ) {
		fprintf(stderr, "\n-- Error: Incorrect first character of pat_samp_mix: %c\n\n", pat_samp_mix);
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	unsigned long seed = atol( argv[14] );

#ifdef DEBUG
	/* DEBUG: Print arguments */
	printf("\nArguments: seq_length=%lu\n", seq_length );
	printf("Arguments: Accumulated probabilitiy G=%f, C=%f, A=%f, T=1\n", prob_G, prob_C, prob_A );
	printf("Arguments: Random patterns number=%d, length_mean=%lu, length_dev=%lu\n", pat_rng_num, pat_rng_length_mean, pat_rng_length_dev );
	printf("Arguments: Sample patterns number=%d, length_mean=%lu, length_dev=%lu, loc_mean=%lu, loc_dev=%lu\n", pat_samp_num, pat_samp_length_mean, pat_samp_length_dev, pat_samp_loc_mean, pat_samp_loc_dev );
	printf("Arguments: Type of mix: %c, Random seed: %lu\n", pat_samp_mix, seed );
	printf("\n");
#endif // DEBUG

        CUDA_CHECK_FUNCTION( hipSetDevice(0) );

	/* 2. Initialize data structures */
	/* 2.1. Skip allocate and fill sequence */
	rng_t random = rng_new( seed );
	rng_skip( &random, seq_length );

	/* 2.2. Allocate and fill patterns */
	/* 2.2.1 Allocate main structures */
	int pat_number = pat_rng_num + pat_samp_num;
	unsigned long *pat_length = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	char **pattern = (char **)malloc( sizeof(char*) * pat_number );
	if ( pattern == NULL || pat_length == NULL ) {
		fprintf(stderr,"\n-- Error allocating the basic patterns structures for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}

	/* 2.2.2 Allocate and initialize ancillary structure for pattern types */
	int ind;
	unsigned long lind;
	#define PAT_TYPE_NONE	0
	#define PAT_TYPE_RNG	1
	#define PAT_TYPE_SAMP	2
	char *pat_type = (char *)malloc( sizeof(char) * pat_number );
	if ( pat_type == NULL ) {
		fprintf(stderr,"\n-- Error allocating ancillary structure for pattern of size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_NONE;

	/* 2.2.3 Fill up pattern types using the chosen mode */
	switch( pat_samp_mix ) {
	case 'A':
		for( ind=0; ind<pat_rng_num; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		break;
	case 'B':
		for( ind=0; ind<pat_samp_num; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		break;
	default:
		if ( pat_rng_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		}
		else if ( pat_samp_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		}
		else if ( pat_rng_num < pat_samp_num ) {
			int interval = pat_number / pat_rng_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_RNG;
				else pat_type[ind] = PAT_TYPE_SAMP;
		}
		else {
			int interval = pat_number / pat_samp_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_SAMP;
				else pat_type[ind] = PAT_TYPE_RNG;
		}
	}

	/* 2.2.4 Generate the patterns */
	for( ind=0; ind<pat_number; ind++ ) {
		if ( pat_type[ind] == PAT_TYPE_RNG ) {
			pattern[ind] = pattern_allocate( &random, pat_rng_length_mean, pat_rng_length_dev, seq_length, &pat_length[ind] );
			generate_rng_sequence( &random, prob_G, prob_C, prob_A, pattern[ind], pat_length[ind] );
		}
		else if ( pat_type[ind] == PAT_TYPE_SAMP ) {
			pattern[ind] = pattern_allocate( &random, pat_samp_length_mean, pat_samp_length_dev, seq_length, &pat_length[ind] );
#define REGENERATE_SAMPLE_PATTERNS
#ifdef REGENERATE_SAMPLE_PATTERNS
			rng_t random_seq_orig = rng_new( seed );
			generate_sample_sequence( &random, random_seq_orig, prob_G, prob_C, prob_A, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#else
			copy_sample_sequence( &random, sequence, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#endif
		}
		else {
			fprintf(stderr,"\n-- Error internal: Paranoic check! A pattern without type at position %d\n", ind );
			exit( EXIT_FAILURE );
		}
	}
	free( pat_type );

	/* Allocate and move the patterns to the GPU */
	unsigned long *d_pat_length;
	char **d_pattern;
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pat_length, sizeof(unsigned long) * pat_number ) );
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pattern, sizeof(char *) * pat_number ) );

	char **d_pattern_in_host = (char **)malloc( sizeof(char*) * pat_number );
	if ( d_pattern_in_host == NULL ) {
		fprintf(stderr,"\n-- Error allocating the patterns structures replicated in the host for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) {
		CUDA_CHECK_FUNCTION( hipMalloc( &(d_pattern_in_host[ind]), sizeof(char *) * pat_length[ind] ) );
        	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern_in_host[ind], pattern[ind], pat_length[ind] * sizeof(char), hipMemcpyHostToDevice ) );
	}
	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern, d_pattern_in_host, pat_number * sizeof(char *), hipMemcpyHostToDevice ) );

	/* Avoid the usage of arguments to take strategic decisions
	 * In a real case the user only has the patterns and sequence data to analize
	 */
	argc = 0;
	argv = NULL;
	pat_rng_num = 0;
	pat_rng_length_mean = 0;
	pat_rng_length_dev = 0;
	pat_samp_num = 0;
	pat_samp_length_mean = 0;
	pat_samp_length_dev = 0;
	pat_samp_loc_mean = 0;
	pat_samp_loc_dev = 0;
	pat_samp_mix = '0';

	/* 2.3. Other result data and structures */
	int pat_matches = 0;

	/* 2.3.1. Other results related to patterns */
	unsigned long *pat_found;
	pat_found = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	if ( pat_found == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux pattern structure for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	
	/* 3. Start global timer */
    CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */

	/* 2.1. Allocate and fill sequence */

	//TODO quitar cuando ya no se necesite
	char *sequence ;

	//2-- Reservar espacio suficiente en la GPU
	char *device_sequence;
	CUDA_CHECK_FUNCTION(hipMalloc(&device_sequence, sizeof(char) * seq_length););
	
	random = rng_new( seed );
	//generate_rng_sequence( &random, prob_G, prob_C, prob_A, sequence, seq_length);

	//3--Generar la sequencia con una funcion Kernel
	// El numero de bloques y su tamaño ( que cada hilo inicialice un de las posiciones de la secuencia)
	// numero de hilos = tamaño de la secuencia
	// numero de grids = numero de threads / 1024
	int blockSize = 1024; //1024 hilos por bloque
	int nunmBlocks;
	if (seq_length % blockSize == 0 ) nunmBlocks = seq_length / blockSize;
	else nunmBlocks = (seq_length / blockSize ) + 1;

	//Usando la memoria reservada para cuda
	new_generate_rng_sequence<<<nunmBlocks, blockSize>>>(random,  prob_G,  prob_C,  prob_A,  device_sequence, seq_length);

#ifdef DEBUG
	/* DEBUG: Print sequence and patterns */
	printf("-----------------\n");
	printf("Sequence: ");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( "%c", sequence[lind] );
	printf("\n-----------------\n");
	printf("Patterns: %d ( rng: %d, samples: %d )\n", pat_number, pat_rng_num, pat_samp_num );
	int debug_pat;
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( "Pat[%d]: ", debug_pat );
		for( lind=0; lind<pat_length[debug_pat]; lind++ ) 
			printf( "%c", pattern[debug_pat][lind] );
		printf("\n");
	}
	printf("-----------------\n\n");
#endif // DEBUG

	/* 2.3.2. Other results related to the main sequence */
	int *seq_matches;
	seq_matches = (int *)malloc( sizeof(int) * seq_length );
	if ( seq_matches == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux sequence structures for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

	/* 4. Initialize ancillary structures */
	for( ind=0; ind<pat_number; ind++) {
		pat_found[ind] = (unsigned long)NOT_FOUND;
	}

	for( lind=0; lind<seq_length; lind++) {
		seq_matches[lind] = 0;
	}

	/* 5. Search for each pattern */

	/* copy to gpu mem pat_found and pat_matches */
	//unsigned long *device_pat_matches;
	int * device_pat_matches;
	int *device_seq_matches;
	unsigned long *device_pat_found;
	unsigned long *device_checksum_found;
	unsigned long *device_checksum_matches;
	unsigned long checksum_found = 0;
	unsigned long checksum_matches =0;

	CUDA_CHECK_FUNCTION(hipMalloc( &device_seq_matches, sizeof(int)*seq_length));
	CUDA_CHECK_FUNCTION(hipMemcpy(device_seq_matches, seq_matches, sizeof(int)*seq_length, hipMemcpyHostToDevice));
	
	CUDA_CHECK_FUNCTION(hipMalloc( &device_pat_matches, sizeof(int)));
	CUDA_CHECK_FUNCTION(hipMemcpy(device_pat_matches, &pat_matches, sizeof(int), hipMemcpyHostToDevice));

	CUDA_CHECK_FUNCTION(hipMalloc(&device_pat_found, sizeof(unsigned long)*pat_number));
	CUDA_CHECK_FUNCTION(hipMemcpy(device_pat_found ,pat_found, sizeof(unsigned long)*pat_number,hipMemcpyHostToDevice));

	CUDA_CHECK_FUNCTION(hipMalloc(&device_checksum_found, sizeof(unsigned long)));
	CUDA_CHECK_FUNCTION(hipMemcpy(device_checksum_found, &checksum_found, sizeof(unsigned long), hipMemcpyHostToDevice));

	CUDA_CHECK_FUNCTION(hipMalloc(&device_checksum_matches, sizeof(unsigned long)));
	CUDA_CHECK_FUNCTION(hipMemcpy(device_checksum_matches, &checksum_matches, sizeof(unsigned long), hipMemcpyHostToDevice));

	// copy the length of each pattern
	CUDA_CHECK_FUNCTION(hipMemcpy(d_pat_length, pat_length, sizeof(unsigned long)*pat_number, hipMemcpyHostToDevice));	

	search_pat_by_gpu<<<nunmBlocks, blockSize>>>(device_sequence, seq_length, pat_number, d_pat_length, d_pattern, device_pat_found, device_seq_matches);
	CUDA_CHECK_KERNEL();

	count_number_of_pat<<<nunmBlocks, blockSize>>>(device_pat_matches, device_pat_found, pat_number, device_checksum_found, seq_length, device_seq_matches, device_checksum_matches);
	CUDA_CHECK_KERNEL();

	//copy the data to the host
	CUDA_CHECK_FUNCTION(hipMemcpy(&pat_matches, device_pat_matches, sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK_FUNCTION(hipMemcpy(&checksum_found, device_checksum_found, sizeof(unsigned long) ,hipMemcpyDeviceToHost));
	CUDA_CHECK_FUNCTION(hipMemcpy(&checksum_matches, device_checksum_matches, sizeof(unsigned long) ,hipMemcpyDeviceToHost));
	
	/* 7. Check sums */
	// calc now the rest operation
	checksum_found = checksum_found % CHECKSUM_MAX;
	checksum_matches= checksum_matches % CHECKSUM_MAX;

#ifdef DEBUG
	/* DEBUG: Write results */
	printf("-----------------\n");
	printf("Found start:");
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( " %lu", pat_found[debug_pat] );
	}
	printf("\n");
	printf("-----------------\n");
	printf("Matches:");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( " %d", seq_matches[lind] );
	printf("\n");
	printf("-----------------\n");
#endif // DEBUG

	/* Free local resources */	
	free( sequence );
	free( seq_matches );

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 8. Stop global timer */
    CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	ttotal = cp_Wtime() - ttotal;

	/* 9. Output for leaderboard */
	printf("\n");
	/* 9.1. Total computation time */
	printf("Time: %lf\n", ttotal );
	
	/* 9.2. Results: Statistics */
	printf("Result: %d, %lu, %lu\n\n", 
			pat_matches,
			checksum_found,
			checksum_matches );
		
	/* 10. Free resources */	
	int i;
	for( i=0; i<pat_number; i++ ) free( pattern[i] );
	free( pattern );
	free( pat_length );
	free( pat_found );

	/* 11. End */
	return 0;
}
